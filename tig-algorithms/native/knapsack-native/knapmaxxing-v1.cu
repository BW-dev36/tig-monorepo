#include "hip/hip_runtime.h"
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <vector>
#include <stdio.h>
#include <iostream>
#include <mutex>
#include <algorithm>
#include "dp_cuda.h"

std::mutex lock_check;
std::mutex race_cond;
//std::lock_guard<std::mutex> lock(lock_check); \

void knapsackCuda(unsigned int *output, const unsigned int *val, const unsigned int *wt, unsigned int W, int num_items, int *selected_indices);

__device__ int maxi(int a, int b) { 
	return (a > b)? a : b; 
}

// __global__ void knapsackKernel(unsigned int *wt, unsigned int *val, unsigned int *output, unsigned int W, int i) {
// 	int w = threadIdx.x;

// 	//__syncthreads();
// 	if (i == 0 || w == 0)
// 		output[(i*W)+w] = 0;
// 	else if (wt[i-1] <= w)
// 		output[(i*W)+w] = maxi(val[i-1] + output[((i-1)*W)+(w-wt[i-1])],  output[((i-1)*W)+w]);
//         printf("Index added %d", )
// 	else
// 		output[(i*W)+w] = output[((i-1)*W)+w];
// 	__syncthreads();
   
// }

__global__ void knapsackKernel(unsigned int *wt, unsigned int *val, unsigned int *output, unsigned int W, int i, int *selected_indices) {
    

    for (int w = threadIdx.x; w <= W; w += blockDim.x)
    {
        if (i == 0 || w == 0) {
            output[(i * (W + 1)) + w] = 0;
            selected_indices[(i * (W + 1)) + w] = -1; // Sentinel value to indicate no item selected
        } else if (wt[i - 1] <= w) {
            unsigned int include_item = val[i - 1] + output[((i - 1) * (W + 1)) + (w - wt[i - 1])];
            unsigned int exclude_item = output[((i - 1) * (W + 1)) + w];
            output[(i * (W + 1)) + w] = maxi(include_item, exclude_item);

            if (include_item > exclude_item) {
                // printf("Index to take : %d\n", i - 1);
                selected_indices[(i * (W + 1)) + w] = i - 1; // Store the index of the included item
            } else {
                selected_indices[(i * (W + 1)) + w] = selected_indices[((i - 1) * (W + 1)) + w]; // Propagate the index of the previous item
            }
        } else {
            output[(i * (W + 1)) + w] = output[((i - 1) * (W + 1)) + w];
            selected_indices[(i * (W + 1)) + w] = selected_indices[((i - 1) * (W + 1)) + w]; // Propagate the index of the previous item
        }
    }
    __syncthreads();
}


int get_nb_gpu()
{
    int nb_gpu = 0;
    hipGetDeviceCount(&nb_gpu);
    return nb_gpu;
}

#include <vector>

extern "C" void solve_challenge_v1_cuda(Challenge challenge, Solution solution) {
    int n = challenge.num_items;
    int W = challenge.max_weight;
    const unsigned int *val = challenge.values;
    const unsigned int *wt = challenge.weights;
    unsigned int *output = 0;
    int *selected_indices = 0;

    output = (unsigned int *)malloc((n + 1) * (W + 1) * sizeof(unsigned int));
    selected_indices = (int *)malloc((n + 1) * (W + 1) * sizeof(int));

    // Create a vector of indices
    std::vector<unsigned int> indices(n);
    for (int i = 0; i < n; ++i) {
        indices[i] = i;
    }

    // Sort indices based on the ratio of value to weight
    std::sort(indices.begin(), indices.end(), [&](int a, int b) {
        return (float)val[a] / wt[a] > (float)val[b] / wt[b];
    });

    // Apply sorted indices to weights and values
    std::vector<unsigned int> sorted_weights(n);
    std::vector<unsigned int> sorted_values(n);
    for (int i = 0; i < n; ++i) {
        sorted_weights[i] = wt[indices[i]];
        sorted_values[i] = val[indices[i]];
    }
    val = sorted_values.data();
    wt = sorted_weights.data();
    // printf("min value %d | max weight %d\n", n, W);
    // printf("input values | ");
    // for (int i = 0; i < n; i++)
    //     printf("%d ", val[i]);
    // printf("|\n");
    // printf("input weight | ");
    // for (int i = 0; i < n; i++)
    //     printf("%d ", wt[i]);
    // printf("|\n");


    knapsackCuda(output, val, wt, W, n, selected_indices);

    // for (int i = 0; i <= n; i++)
	// 	for (int j = 0; j <= W; j++) {
	// 		std::cout << output[i*(W + 1) + j] << ";";
	// 		if (j == W)
	// 			std::cout << std::endl;
	// }

    //std::cout << "Maximum Value possible for knapsack with capacity " << W << " is : " << output[(n +1) * (W + 1) - 1] << std::endl;

    // Retrieve the indices of the selected items
    int currentW = W;
    int count = 0;

    std::vector<bool> taken(n, false);
    // printf("Last colomuns : ");

    // for (int i = 1; i <= n; i++) {
    //     int value = output[i * (W + 1) + W - 1] - output[(i - 1) * (W + 1) + W - 1];

    //     //Find value corresponding with the lightest w
    //     printf("%d ", value);
    // }
    // printf("\n");

    for (int i = n; i > 0 ; --i) {

        if ((i) * (W + 1) + currentW >= (n + 1) * (W + 1))
        {
            printf("Out of bound index (i) * (W + 1) + currentW\n");
        }

        int idx = selected_indices[(i) * (W + 1) + currentW];

        if (idx >= n)
        {
            printf("Out of bound index Idx : %d : %d %d \n", (i) * (W + 1) + currentW, i, currentW);
        }


        if (idx != -1 && !taken[idx]) {
            solution.items[count++] = indices[idx];
            taken[idx] = true;
            currentW -= wt[idx];
            if (currentW < 0) currentW = 0;
            // Update the indices to avoid duplicate selections
            for (int j = i - 1; j >= 0; --j) {
                
                if (selected_indices[j * (W + 1) + currentW] == idx) {
                    selected_indices[j * (W + 1) + currentW] = -1;
                }
            }
        }
        // else if (idx != -1 && taken[idx]) // Find the next one
        // {
        //     int value = val[idx];
        //     int int_max = 99999;
        //     int best_weight = int_max;
        //     int best_index  = int_max;
        //     for (int k = 0; k < n; k++)
        //     {
        //         if (value == val[idx] && !taken[k] && best_weight > wt[idx])
        //         {
        //             best_index = k;
        //             best_weight = wt[idx];
        //             taken[best_index] = true;
        //             solution.items[count++] = best_index;
        //             currentW -= wt[best_index];
        //             break;
        //         }
        //     }
        //     // if (best_index != int_max)
        //     // {
        //     //     taken[best_index] = true;
        //     //     solution.items[count++] = best_index;
        //     //     currentW -= wt[best_index];
        //     // }
        // }
    }


    // //Print indices and calculate final value and weight
    // printf("Indices |\n");
    // int final_value = 0;
    // int final_weight = 0;
    // for (int i = 0; i < count; ++i) {
    //     int idx = solution.items[i];
    //     final_value += val[idx];
    //     final_weight += wt[idx];
    //     printf("Indice = %d, (%d %d) => Sum => %d %d\n", idx, val[idx], wt[idx], final_value, final_weight);
    // }

    // if (final_value != output[n * (W + 1) + W]) {
    //     printf("Invalid solution: Total value (%d) does not reach maximum value (%d)\n", final_value, output[(n + 1) * (W + 1)]);
    // }

    // if (final_weight > W) {
    //     printf("Invalid solution: Total weight (%d) exceeded max weight (%d)\n", final_weight, W);
    // }


    free(output);
    free(selected_indices);

}

#include <thread>
#include <atomic>
// Global variable to track the next GPU to assign
std::atomic<unsigned int> next_gpu_index(0);

void knapsackCuda(unsigned int *output, const unsigned int *val, const unsigned int *wt, unsigned int W, int num_items, int *selected_indices) {
    unsigned int *dev_val = 0;
    unsigned int *dev_wt = 0;
    unsigned int *dev_output = 0;
    int *dev_selected_indices = 0;

    std::lock_guard<std::mutex> lock(lock_check);
	// hipEvent_t start, stop;
	// hipEventCreate(&start);
	// hipEventCreate(&stop);
    //std::lock_guard<std::mutex> lock(local_guard);
    unsigned int gpu_to_use = (next_gpu_index++) % get_nb_gpu();
    CUDA_CHECK(hipSetDevice(gpu_to_use));

    hipStream_t stream;
    hipStreamCreate(&stream);

    CUDA_CHECK(hipMalloc((void**)&dev_output, (num_items + 1) * (W + 1) * sizeof(unsigned int)));
    

    CUDA_CHECK(hipMalloc((void**)&dev_val, num_items * sizeof(unsigned int)));
    

    CUDA_CHECK(hipMalloc((void**)&dev_wt, num_items * sizeof(unsigned int)));
    
    
    CUDA_CHECK(hipMalloc((void**)&dev_selected_indices, (num_items + 1) * (W + 1) * sizeof(int)));
    

    CUDA_CHECK(hipMemcpy(dev_val, val, num_items * sizeof(unsigned int), hipMemcpyHostToDevice));

    CUDA_CHECK(hipMemcpy(dev_wt, wt, num_items * sizeof(unsigned int), hipMemcpyHostToDevice));

	//hipEventRecord(start);
	// Determine number of threads per block
    int threadsPerBlock = 128;

    // Launch a kernel on the GPU with one block per iteration
    {
        //std::cout << "Thread Id : " << std::this_thread::get_id() << std::endl;     
        for (int i = 0; i <= num_items; i++) {
            knapsackKernel<<<1, threadsPerBlock,0, stream>>>(dev_wt, dev_val, dev_output, W, i, dev_selected_indices);
        }
    }
    CUDA_CHECK(hipStreamSynchronize(stream));

    // CUDA_CHECK(hipDeviceSynchronize());
	//hipEventRecord(stop);

    CUDA_CHECK(hipMemcpy(output, dev_output, (num_items + 1) * (W + 1) * sizeof(unsigned int), hipMemcpyDeviceToHost));

    // Copy selected indices from GPU buffer to host memory.
    CUDA_CHECK(hipMemcpy(selected_indices, dev_selected_indices, (num_items + 1) * (W + 1) * sizeof(int), hipMemcpyDeviceToHost));

	//hipEventSynchronize(stop);
	//float milliseconds = 0;
	//hipEventElapsedTime(&milliseconds, start, stop);

	//std::cout << "Execution Time : " << milliseconds / 1000 << " seconds" << std::endl;

    CUDA_CHECK(hipFree(dev_output));
    CUDA_CHECK(hipFree(dev_val));
    CUDA_CHECK(hipFree(dev_wt));
    CUDA_CHECK(hipFree(dev_selected_indices));
    CUDA_CHECK(hipStreamDestroy(stream));
}