#include "hip/hip_runtime.h"
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <vector>
#include <stdio.h>
#include <iostream>
#include <mutex>
#include <algorithm>
#include <thread>
#include <atomic>

#include "utils.h"

static std::mutex lock_check;
//std::lock_guard<std::mutex> lock(lock_check); \

#define GENERAL_MAX_WEIGHT 10000
#define GENERAL_MAX_NUM_ITEMS 150

// Global variable to track the next GPU to assign
static std::atomic<unsigned int> next_gpu_index(0);
static std::atomic<unsigned int> next_workspace_index(0);

extern "C" {
    typedef struct {
        unsigned int max_weight;
        unsigned int min_value;
        unsigned int num_items;
        const unsigned int* values;
        const unsigned int* weights;
    } Challenge;

    typedef struct {
        int* items;
    } Solution;

    void solve_challenge_cuda(Challenge challenge, Solution solution);
}

class Workspace {
    public:
        unsigned int * dp;

        unsigned int * d_values;
        unsigned int * d_weights;
        unsigned int * d_indices;
        unsigned int * d_dp;
        int           num_items;
        int           max_weight;
        hipStream_t stream;
        


        std::atomic<int> in_use;
        int gpu_to_use;

        Workspace() : in_use(0) {
            InitDeviceAllocation();
        }

        void InitDeviceAllocation()
        {
            gpu_to_use = (next_gpu_index++) % get_nb_gpu();

            CUDA_CHECK(hipSetDevice(gpu_to_use));

            hipStreamCreate(&stream);

            dp = (unsigned int *)malloc((GENERAL_MAX_NUM_ITEMS + 1) * (GENERAL_MAX_WEIGHT + 1) * sizeof(unsigned int));

            CUDA_CHECK(hipMallocAsync((void**)&d_dp, (GENERAL_MAX_NUM_ITEMS + 1) * (GENERAL_MAX_WEIGHT + 1) * sizeof(unsigned int), stream));
            

            CUDA_CHECK(hipMallocAsync((void**)&d_values, GENERAL_MAX_NUM_ITEMS * sizeof(unsigned int), stream));
            

            CUDA_CHECK(hipMallocAsync((void**)&d_weights, GENERAL_MAX_NUM_ITEMS * sizeof(unsigned int), stream));
   
        }

        void initializeWorkspace(Challenge & challenge)
        {
            num_items = challenge.num_items;
            max_weight = challenge.max_weight;

            CUDA_CHECK(hipSetDevice(gpu_to_use));

            CUDA_CHECK(hipMemcpyAsync(d_values, challenge.values, num_items * sizeof(unsigned int), hipMemcpyHostToDevice, stream));

            CUDA_CHECK(hipMemcpyAsync(d_weights, challenge.weights, num_items * sizeof(unsigned int), hipMemcpyHostToDevice, stream));
        }



        unsigned int * retrieve_dp()
        {
            CUDA_CHECK(hipStreamSynchronize(stream));
            CUDA_CHECK(hipMemcpyAsync(dp, d_dp, (num_items + 1) * (max_weight + 1) * sizeof(unsigned int), hipMemcpyDeviceToHost, stream));

            return dp;
        }

        ~Workspace()
        {
            CUDA_CHECK(hipFreeAsync(d_dp, stream));
            CUDA_CHECK(hipFreeAsync(d_values, stream));
            CUDA_CHECK(hipFreeAsync(d_weights, stream));
            CUDA_CHECK(hipStreamDestroy(stream));
            free(dp);
        }
};

static std::once_flag init_flag;

static const int nb_workspace = 128;
static std::vector<Workspace *> * workspaces = nullptr;

static void initWorkspace()
{
    std::vector<Workspace *> *l_workspaces = new std::vector<Workspace *>(nb_workspace);

    std::thread::id thread_id = std::this_thread::get_id();
    std::cout << "ThreadId = " << thread_id << " ==> Initialize workspace..." << std::endl;
    
    for (int i = 0; i < nb_workspace; i++) 
    {
        Workspace* workspace_selected = new Workspace();
        (*l_workspaces)[i] = workspace_selected;

    }
    std::cout << "ThreadId = " << thread_id << " ==> Initialize workspace OK" << std::endl;
    workspaces = l_workspaces;
}

__device__ int maxi(int a, int b) { 
	return (a > b)? a : b; 
}


__global__ void knapsackKernel(unsigned int *wt, unsigned int *val, unsigned int *dp, unsigned int W, int i, int item_index) {
        int item_weight = wt[item_index];
        int item_value = val[item_index];
        const int max_weight_plus_one = W + 1;

        for (int w = W - (blockIdx.x * blockDim.x + threadIdx.x); w >= item_weight; w -= gridDim.x * blockDim.x)
        {
            int prev_state = (i - 1) * max_weight_plus_one + w;
            int curr_state = i * max_weight_plus_one + w;
            dp[curr_state] = maxi(dp[prev_state], dp[prev_state - item_weight] + item_value);
        }
}


#include <vector>

// Comparator function for sorting by value/weight ratio
static bool compareItems(const std::pair<int, double>& a, const std::pair<int, double>& b) {
    return a.second > b.second;
}

extern "C" void solve_knapmaxxing_v2_cuda(Challenge challenge, Solution solution) {
    int n = challenge.num_items;
    int W = challenge.max_weight;

    int total_w = 0;
    int total_v = 0;
    for (int i = 0; i < n; i++)
    {
        total_w += challenge.weights[i];
        total_v += challenge.values[i];
    }
    if (total_w <= W && total_v >= challenge.min_value)
    {
        for (int i = 0; i < n; i++)
        {
            solution.items[i] = i;
        }
        return;
    }
    
    // Sort items by value/weight ratio (descending order)
    std::vector<std::pair<int, double>> sorted_items(n);
    for (int i = 0; i < n; ++i) {
        sorted_items[i] = std::make_pair(i, (double)challenge.values[i] / challenge.weights[i]);
    }
    std::sort(sorted_items.begin(), sorted_items.end(), compareItems);
    
  
    // printf("min value %d | max weight %d\n", n, W);
    // printf("input values | ");
    // for (int i = 0; i < n; i++)
    //     printf("%d ", val[i]);
    // printf("|\n");
    // printf("input weight | ");
    // for (int i = 0; i < n; i++)
    //     printf("%d ", wt[i]);
    // printf("|\n");

    // Calculate upper bound
    int upper_bound = 0;
    int remaining_weight = W;
    for (const auto& [item_index, ratio] : sorted_items) {
        int item_weight = challenge.weights[item_index];
        int item_value = challenge.values[item_index];

        if (item_weight <= remaining_weight) {
            upper_bound += item_value;
            remaining_weight -= item_weight;
        } else {
            upper_bound += (int)std::floor(ratio * remaining_weight);
            break;
        }
    }

    if (upper_bound < challenge.min_value) {
        solution.items[0] = -2; 
        return;
    }

    std::thread::id thread_id = std::this_thread::get_id();
    std::call_once(init_flag, initWorkspace);
    //std::cout << "ThreadId = " << thread_id << " ==> choosing workspace... " <<  std::endl;
    // while (workspaces == nullptr)
    // {
    //     std::call_once(init_flag, initWorkspace);
    //     std::this_thread::sleep_for(std::chrono::seconds(2));
    // }

    int workspace_id = -1; 
    Workspace *workspace_ptr = nullptr;
    while (workspace_ptr == nullptr)
    {
        int expected = 0;
        workspace_id = (next_workspace_index++) % nb_workspace;
        
        if ((*workspaces)[workspace_id]->in_use.compare_exchange_strong(expected, 1))
        {
            workspace_ptr = (*workspaces)[workspace_id];
            break;
        }
    }
   
    Workspace &workspace = *workspace_ptr;

    //std::cout << "ThreadId = " << thread_id << " ==> Choose Workspace Id = " << workspace_id << " GPU ID = " << workspace.gpu_to_use <<  std::endl;
    
    workspace.initializeWorkspace(challenge);

    
    // Dynamic Programming (DP)
    const int max_weight_plus_one = challenge.max_weight + 1;
    const int num_states = (challenge.num_items + 1) * max_weight_plus_one;
    for (int i = 1; i <= challenge.num_items; ++i) {
       const auto& [item_index, _] = sorted_items[i - 1];

       knapsackKernel<<<4, 256, 0, workspace.stream>>>(workspace.d_weights, workspace.d_values, workspace.d_dp, workspace.max_weight, i, item_index); 
    }
    unsigned int *dp = workspace.retrieve_dp();
    
    if (dp[challenge.max_weight] < challenge.min_value) {
        solution.items[0] = -2;
        workspace.in_use = 0;
        return ;
    }

    
    // 5. Récupération de la solution
    unsigned int i = challenge.num_items;
    int w = challenge.max_weight;
    int total_value = 0;
    int solution_index = 0; 
    {
        //std::lock_guard<std::mutex> lock(lock_check);

        while (i > 0 && total_value < challenge.min_value) {
            unsigned int prev_state = (i - 1) * (challenge.max_weight + 1) + w;
            unsigned int curr_state = i * (challenge.max_weight + 1) + w;
            const auto& [item_index, _] = sorted_items[i - 1];
            unsigned int item_weight = challenge.weights[item_index];
            unsigned int item_value = challenge.values[item_index];
            //printf("ThreadId = %lu ==> i = %d w = %d   item_weight %d\n", thread_id, i, w, item_weight);
            if (dp[curr_state] != dp[prev_state]) {
                solution.items[solution_index++] = item_index;
                w -= item_weight;
                
                if (w <= 0)  {
                    //printf("Break that hell %d\n", w);
                    break;
                }

                total_value += item_value;
            }
            
            i--;
        }
    }
    workspace.in_use = 0;
    if (solution_index > 0)
    {
        std::cout << "workspace ID :" << workspace_id << "GPU : " << workspace.gpu_to_use << " found potential solution" << std::endl;
    }
    else 
    {
        std::cout << "workspace ID :" << workspace_id << "GPU : " << workspace.gpu_to_use << " no solution" << std::endl;
    }
    
}
