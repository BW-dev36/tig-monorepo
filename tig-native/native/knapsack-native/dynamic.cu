#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <algorithm>
#include <vector>
#include "utils.h"
#include "dp_cuda.h"

__device__ float calculate_ratio(unsigned int value, unsigned int weight) {
    return static_cast<float>(value) / static_cast<float>(weight);
}

__global__ void sort_items_kernel(unsigned int* d_values, unsigned int* d_weights, int* d_indices, int num_items) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_items - 1) {
        float ratio_a = calculate_ratio(d_values[d_indices[idx]], d_weights[d_indices[idx]]);
        float ratio_b = calculate_ratio(d_values[d_indices[idx+1]], d_weights[d_indices[idx+1]]);
        if (ratio_b > ratio_a) {
            int temp = d_indices[idx];
            d_indices[idx] = d_indices[idx+1];
            d_indices[idx+1] = temp;
        }
    }
}

__global__ void solve_knapsack_kernel(unsigned int* d_values, unsigned int* d_weights, int* d_indices, 
                                      int* d_solution, unsigned int max_weight, unsigned int min_value, 
                                      int num_items, int* d_found_solution) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < (1 << num_items)) {
        unsigned int current_weight = 0;
        unsigned int current_value = 0;
        for (int i = 0; i < num_items; ++i) {
            if (idx & (1 << i)) {
                int item_idx = d_indices[i];
                current_weight += d_weights[item_idx];
                current_value += d_values[item_idx];
            }
        }
        if (current_weight <= max_weight && current_value >= min_value) {
            atomicExch(d_found_solution, 1);
            for (int i = 0; i < num_items; ++i) {
                if (idx & (1 << i)) {
                    printf("Solution Found !!");
                    d_solution[d_indices[i]] = 1;
                } else {
                    d_solution[d_indices[i]] = 0;
                }
            }
        }
    }
}

extern "C" void solve_dynamic_cuda(Challenge challenge, Solution solution) {
    int num_items = challenge.num_items;
    unsigned int max_weight = challenge.max_weight;
    unsigned int min_value = challenge.min_value;

    // Allocate device memory
    unsigned int *d_values, *d_weights;
    int *d_indices, *d_solution, *d_found_solution;

    CUDA_CHECK(hipMalloc(&d_values, num_items * sizeof(unsigned int)));
    CUDA_CHECK(hipMalloc(&d_weights, num_items * sizeof(unsigned int)));
    CUDA_CHECK(hipMalloc(&d_indices, num_items * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_solution, num_items * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_found_solution, sizeof(int)));

    // Copy data to device
    CUDA_CHECK(hipMemcpy(d_values, challenge.values, num_items * sizeof(unsigned int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_weights, challenge.weights, num_items * sizeof(unsigned int), hipMemcpyHostToDevice));

    // Initialize indices
    int* h_indices = new int[num_items];
    for (int i = 0; i < num_items; ++i) {
        h_indices[i] = i;
    }
    CUDA_CHECK(hipMemcpy(d_indices, h_indices, num_items * sizeof(int), hipMemcpyHostToDevice));
    delete[] h_indices;

    // Sort items by value-to-weight ratio
    int block_size = 256;
    int grid_size = (num_items + block_size - 1) / block_size;
    for (int i = 0; i < num_items; ++i) {
        sort_items_kernel<<<grid_size, block_size>>>(d_values, d_weights, d_indices, num_items);
        CUDA_CHECK(hipDeviceSynchronize());
    }

    // Initialize solution
    CUDA_CHECK(hipMemset(d_solution, 0, num_items * sizeof(int)));
    CUDA_CHECK(hipMemset(d_found_solution, 0, sizeof(int)));

    // Solve knapsack problem
    int num_combinations = 1 << num_items;
    grid_size = (num_combinations + block_size - 1) / block_size;
    solve_knapsack_kernel<<<grid_size, block_size>>>(d_values, d_weights, d_indices, d_solution, 
                                                     max_weight, min_value, num_items, d_found_solution);
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy solution back to host
    CUDA_CHECK(hipMemcpy(solution.items, d_solution, num_items * sizeof(int), hipMemcpyDeviceToHost));

    // Clean up
    CUDA_CHECK(hipFree(d_values));
    CUDA_CHECK(hipFree(d_weights));
    CUDA_CHECK(hipFree(d_indices));
    CUDA_CHECK(hipFree(d_solution));
    CUDA_CHECK(hipFree(d_found_solution));
}